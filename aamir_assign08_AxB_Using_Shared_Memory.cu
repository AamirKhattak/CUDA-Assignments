#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include< iostream>
#include <ctime>
using namespace std;

/* 
#CUDA PROGRAM STRUCTURE

1. Memory Allocation on CPU and GPU
2. Initialization of Memory in CPU
3. Memcpy to GPU
4. Kernel Invocation
5. Memcpy to CPU
*/
//----------------[START] CUDA KERNEL CODE ---------------------------

const int TILE_WIDTH = 32;

__global__ void MulKernel(int *A, int *B, int *C,int WIDTH, int HEIGHT)
{
	//const int TILE_WIDTH = 32;
	//const int Mat_SIZE = WIDTH * HEIGHT;

	//int col = ( blockDim.x * blockIdx.x) + threadIdx.x;
	//int row = ( blockDim.y * blockIdx.y) + threadIdx.y;

	////int index = col + ( WIDTH * row);



	__shared__ float d_A [ TILE_WIDTH] [TILE_WIDTH];
	__shared__ float d_B [ TILE_WIDTH] [TILE_WIDTH];

	int bx = blockIdx.x;
	int by = blockIdx.y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;

	int value=0;

	for( int m =0 ; m<WIDTH/TILE_WIDTH; ++m){

		d_A[ty][tx] = A[Row * WIDTH + m * TILE_WIDTH + tx];
		d_B[ty][tx] = B[ Col + (m*TILE_WIDTH+ty)*WIDTH];

		__syncthreads();

		for( int k =0; k<TILE_WIDTH; ++k)
			value += d_A[ty][k] * d_B[k][tx];
		__syncthreads();
	}

	if( Row < HEIGHT && Col < WIDTH){
		int value = 0;
		// will run calculate product for a pixel
		for(int i = 0; i<WIDTH; i++){
			value += A[ (Row * HEIGHT) + i ] * B [ (i * WIDTH) + Col ];
		}		

		C[Row*WIDTH+Col] = value;
	}

}
//---------------- [END] CUDA KERNEL CODE ----------------------------



//will show the memory used by one array i.e a/b/c
// viewMemoryUse( WIDTH, HEIGHT) : specific to this program
void viewMemUse(int, int);
// IniArray( ARRAY, WIDTH, HEIGHT, RandomValueSeed)
void initializeArray(int*, int, int, int);
// DisplayArray( arrayNAme i.e H_A, array, width, height)
void displayArray(char*, int *,int,int);
//
void mulKernelCPU( int*, int*, int*, int, int);
//
void compareResult( int *arrayA, int *arrayB, int width, int height);

int main()
{
	const int WIDTH = 2048;
	const int HEIGHT = 2048;

	int *h_a, *h_b, *h_c;
	int *d_a, *d_b, *d_c;

	//## 1. Memory Allocation on HOST & DEVICE
	//1.a Memory allocation on HOST
	int SIZE_IN_BYTES = WIDTH * HEIGHT * sizeof(int);
	h_a = (int *) malloc( SIZE_IN_BYTES); // since square matrix so  A = [ WIDTH * WIDTH]
	h_b = (int *) malloc( SIZE_IN_BYTES);
	h_c = (int *) malloc( SIZE_IN_BYTES);

	//1.b Memory Allocation on DEVICE
	hipMalloc( (void **) &d_a, SIZE_IN_BYTES);
	hipMalloc( (void **) &d_b, SIZE_IN_BYTES);
	hipMalloc( (void **) &d_c, SIZE_IN_BYTES);

	//## 2. Memory Initialization HOST
	//Initializing Host Arrays
	initializeArray( h_a, WIDTH, HEIGHT, 50);
	initializeArray( h_b, WIDTH, HEIGHT, 30);

	//## 3. Memcpy HOST to DEVICE
	hipMemcpy( d_a, h_a, SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy( d_b, h_b, SIZE_IN_BYTES, hipMemcpyHostToDevice);

	//## 4. Kernel Invocation
	int mat_size= WIDTH * HEIGHT;
	int threadX = 32;
	int threadY = 32;
	int blockX = ceil( WIDTH/threadX) +1;
	int blockY = ceil( HEIGHT/threadY) +1;

	dim3 dimBlock( threadX, threadY, 1);
	dim3 dimGrid( blockX, blockY, 1);

	MulKernel<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, WIDTH, HEIGHT);

	////## 5. Memcpy DEVICE to HOST
	hipMemcpy( h_c, d_c, SIZE_IN_BYTES, hipMemcpyDeviceToHost);


	// for comparing results
	int *cpu_results; // to store CPU results
	cpu_results = (int *) malloc( SIZE_IN_BYTES);
	//mulKernelCPU( h_a, h_b, cpu_results, WIDTH, HEIGHT);

	// Displaying Result
	cout<<"Comparing and Displaying Result"<<endl;
	//compareResult( h_c, cpu_results, WIDTH, HEIGHT);

	cout<<endl<<"Showing some data : "<<endl;
	displayArray( "a",h_a, 5, 5);
	displayArray("b", h_b, 5, 5);
	displayArray("c",h_c, 5, 5);
	//displayArray("cpu_result",cpu_results,2,2);

	hipFree(&d_a);
	hipFree(&d_b);
	hipFree(&d_c);

	free(h_a);
	free(h_b);
	free(h_c);


	system("pause");
	return 0;
}

/*void mulKernelCPU( int *arrayA, int *arrayB, int *arrayC, int width, int height){
int arraySize = width * height;
for(int i=0; i<arraySize; i++){
int value;
for( int j=0; j<arraySize; j++){
value = arrayA[j * height + j ] * arrayB[i * width + i];	
}
int index = 0;
arrayC[ index ] = value;
}
}*/

/*
value = A[ (row * HEIGHT) + i ] * B [ (i * WIDTH) + col ];
}		
C[index] = value;
*/

void compareResult( int *arrayA, int *arrayB, int width, int height){

	int arraySize = width * height;

	for(int i=0; i<arraySize; i++){
		if( arrayA[i] != arrayB[i]){
			cout<<"arrayA["<<i<<"] != arrayB["<<i<<"]"<<endl;
			cout<<"[NOT SAME] Result on CPU and GPU is not same"<<endl;
			break;
		}
		if (i ==arraySize-1){
			cout<<"Result on CPU and GPU is same"<<endl;
		}
	}
}

void initializeArray(int *array, int width, int height, int randomValueSEED){
	int MAT_SIZE = width * height;
	// Initializing Array with random values
	srand ( time(NULL) );	
	for( int i=0; i<MAT_SIZE; i++){
		int value = rand() % randomValueSEED + 1;
		array[i] = value;
	}
}

void displayArray(char* arrayName,int *array,int width, int height){
	cout<<"Displaying Values of Array: "<<arrayName<<endl;
	for(int i=0; i<width*height; i++){	
		if( i % width == 0)
			cout<<endl;
		//cout<<"Array["<<i<<"] : "<<array[i]<<"  ";
		cout<<"["<<i<<"] : "<<array[i]<<"  ";
	}
	cout<<endl;
}

void viewMemUse(int pWidth, int pHeight){
	int size = pWidth * pHeight * sizeof(int);
	cout<<"Size: of WIDTH * HEIGHT * sizeof(int)"<<endl;
	cout<<"Size = "<<pWidth<<" * "<<pHeight<<" * sizeof(int)"<<endl;
	cout<<"Size: BYTES "<<size<<endl;
	cout<<"Size: KBYTES "<<size/1024<<endl;
	cout<<"Size: MBYTES "<<(size/1024)/1024<<endl;
	float gSize = ((size/1024.0)/1024.0)/1024.0;
	cout<<"Size: GBYTES "<<gSize<<endl;

}
